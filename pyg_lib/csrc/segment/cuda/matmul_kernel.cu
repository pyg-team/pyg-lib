#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <torch/library.h>

#include <cutlass/cutlass.h>
#include <cutlass/gemm/device/gemm_grouped.h>
#include <cutlass/gemm/device/gemm_universal.h>
#include <cutlass/gemm/gemm.h>
#include <cutlass/gemm/kernel/default_gemm_grouped.h>
#include <cutlass/gemm/kernel/gemm_grouped.h>
#include <cutlass/util/command_line.h>
#include <cutlass/util/device_memory.h>
#include <cutlass/util/distribution.h>
#include <cutlass/util/host_tensor.h>
#include <cutlass/util/reference/device/gemm_complex.h>
#include <cutlass/util/reference/device/tensor_fill.h>
#include <cutlass/util/reference/host/gemm_complex.h>
#include <cutlass/util/reference/host/tensor_compare.h>
#include <cutlass/util/reference/host/tensor_copy.h>
#include <cutlass/util/reference/host/tensor_norm.h>
#include <cutlass/util/tensor_view_io.h>

namespace pyg {
namespace segment {

namespace {

at::Tensor matmul_kernel(const at::Tensor& input,
                         const at::Tensor& ptr,
                         const at::Tensor& other,
                         const at::Tensor& out) {
  // TODO: Require contiguous memory!
  auto num_matrices = ptr.numel() - 1;

  using GemmKernel = typename cutlass::gemm::kernel::DefaultGemmGrouped<
      float,                                                          //
      cutlass::layout::RowMajor,                                      //
      cutlass::ComplexTransform::kNone,                               //
      8,                                                              //
      float,                                                          //
      cutlass::layout::RowMajor,                                      //
      cutlass::ComplexTransform::kNone,                               //
      8,                                                              //
      float,                                                          //
      cutlass::layout::RowMajor,                                      //
      float,                                                          //
      cutlass::arch::OpClassTensorOp,                                 //
      cutlass::arch::Sm80,                                            //
      cutlass::gemm::GemmShape<256, 128, 32>,                         //
      cutlass::gemm::GemmShape<64, 64, 32>,                           //
      cutlass::gemm::GemmShape<16, 8, 8>,                             //
      cutlass::epilogue::thread::LinearCombination<                   //
          float, 8, float, float>,                                    //
      cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<8>,  //
      2,                                                              //
      cutlass::arch::OpMultiplyAdd                                    //
      >::GemmKernel;

  auto ptr_data = ptr.cpu().data_ptr<int64_t>();

  std::vector<float*> ptr_A_host(num_matrices);
  std::vector<float*> ptr_B_host(num_matrices);
  std::vector<float*> ptr_D_host(num_matrices);

  for (size_t i = 0; i < num_matrices; ++i) {
    ptr_A_host[i] = input.data_ptr<float>() + (ptr_data[i] * input.size(1));
    ptr_B_host[i] = other[i].data_ptr<float>();
    ptr_D_host[i] = out.data_ptr<float>() + (ptr_data[i] * out.size(1));
  }

  cutlass::DeviceAllocation<float*> ptr_A;
  ptr_A.reset(num_matrices);
  ptr_A.copy_from_host(ptr_A_host.data());

  cutlass::DeviceAllocation<float*> ptr_B;
  ptr_B.reset(num_matrices);
  ptr_B.copy_from_host(ptr_B_host.data());

  cutlass::DeviceAllocation<float*> ptr_D;
  ptr_D.reset(num_matrices);
  ptr_D.copy_from_host(ptr_D_host.data());

  std::vector<cutlass::gemm::GemmCoord> all_problems(num_matrices);
  std::vector<int64_t> lda_host(num_matrices);
  std::vector<int64_t> ldb_host(num_matrices);
  std::vector<int64_t> ldd_host(num_matrices);
  for (size_t i = 0; i < num_matrices; ++i) {
    auto m = ptr_data[i + 1] - ptr_data[i];
    auto k = input.size(1);
    auto n = out.size(1);
    all_problems[i] = cutlass::gemm::GemmCoord(m, n, k);
    lda_host[i] = GemmKernel::LayoutA::packed({m, k}).stride(0);
    ldb_host[i] = GemmKernel::LayoutB::packed({k, n}).stride(0);
    ldd_host[i] = GemmKernel::LayoutC::packed({m, n}).stride(0);
  }

  cutlass::DeviceAllocation<cutlass::gemm::GemmCoord> all_problems_device;
  all_problems_device.reset(num_matrices);
  all_problems_device.copy_from_host(all_problems.data());

  cutlass::DeviceAllocation<int64_t> lda;
  lda.reset(num_matrices);
  lda.copy_from_host(lda_host.data());

  cutlass::DeviceAllocation<int64_t> ldb;
  ldb.reset(num_matrices);
  ldb.copy_from_host(ldb_host.data());

  cutlass::DeviceAllocation<int64_t> ldd;
  ldd.reset(num_matrices);
  ldd.copy_from_host(ldd_host.data());

  /* configurate the GEMM args */
  using EpilogueOutputOp = typename GemmKernel::Epilogue::OutputOp;
  typename EpilogueOutputOp::Params epilogue_op(1.0, 0.0);

  using GemmGrouped = cutlass::gemm::device::GemmGrouped<GemmKernel>;
  int threadblock_count = 0;
  typename GemmGrouped::Arguments args(
      all_problems_device.get(), num_matrices, threadblock_count, epilogue_op,
      ptr_A.get(), ptr_B.get(), ptr_D.get(), ptr_D.get(), lda.get(), ldb.get(),
      ldd.get(), ldd.get());

  GemmGrouped gemm;
  cutlass::Status status;
  status = gemm.initialize(args);
  TORCH_CHECK(status == cutlass::Status::kSuccess,
              "GroupedGEMM kernel initialization: failed \n");
  status = gemm.run();
  TORCH_CHECK(status == cutlass::Status::kSuccess,
              "GroupedGEMM kernel run: failed \n");

  return out;
}

}  // namespace

TORCH_LIBRARY_IMPL(pyg, CUDA, m) {
  m.impl(TORCH_SELECTIVE_NAME("pyg::segment_matmul"), TORCH_FN(matmul_kernel));
}

}  // namespace segment
}  // namespace pyg
