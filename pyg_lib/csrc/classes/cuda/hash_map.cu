#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <torch/library.h>
#include <limits>

#ifndef _WIN32
#include <cuco/static_map.cuh>
#endif

namespace pyg {
namespace classes {

namespace {

#define DISPATCH_CASE_KEY(...)                         \
  AT_DISPATCH_CASE(at::ScalarType::Short, __VA_ARGS__) \
  AT_DISPATCH_CASE(at::ScalarType::Int, __VA_ARGS__)   \
  AT_DISPATCH_CASE(at::ScalarType::Long, __VA_ARGS__)

#define DISPATCH_KEY(TYPE, NAME, ...) \
  AT_DISPATCH_SWITCH(TYPE, NAME, DISPATCH_CASE_KEY(__VA_ARGS__))

struct HashMapImpl {
  virtual ~HashMapImpl() = default;
  virtual at::Tensor get(const at::Tensor& query) = 0;
  virtual at::Tensor keys() = 0;
  virtual int64_t size() = 0;
  virtual at::ScalarType dtype() = 0;
  virtual at::Device device() = 0;
};

#ifndef _WIN32
template <typename KeyType>
struct CUDAHashMapImpl : HashMapImpl {
 public:
  using ValueType = int64_t;

  CUDAHashMapImpl(const at::Tensor& key, double load_factor)
      : device_(key.device()) {
    c10::cuda::MaybeSetDevice(key.get_device());

    KeyType constexpr empty_key_sentinel = std::numeric_limits<KeyType>::min();
    ValueType constexpr empty_value_sentinel = -1;

    size_t capacity = std::ceil(key.numel() / load_factor);
    map_ = std::make_unique<cuco::static_map<KeyType, ValueType>>(
        capacity, cuco::empty_key{empty_key_sentinel},
        cuco::empty_value{empty_value_sentinel});

    const auto key_data = key.data_ptr<KeyType>();
    const auto options =
        key.options().dtype(c10::CppTypeToScalarType<ValueType>::value);
    const auto value = at::arange(key.numel(), options);
    const auto value_data = value.data_ptr<ValueType>();
    const auto zipped =
        thrust::make_zip_iterator(thrust::make_tuple(key_data, value_data));

    map_->insert(zipped, zipped + key.numel());
  }

  at::Tensor get(const at::Tensor& query) override {
    c10::cuda::MaybeSetDevice(query.get_device());

    const auto options =
        query.options().dtype(c10::CppTypeToScalarType<ValueType>::value);
    const auto out = at::empty({query.numel()}, options);
    const auto query_data = query.data_ptr<KeyType>();
    const auto out_data = out.data_ptr<ValueType>();

    map_->find(query_data, query_data + query.numel(), out_data);

    return out;
  }

  at::Tensor keys() override {
    c10::cuda::MaybeSetDevice(device_.index());

    const auto options = at::TensorOptions().device(device_);
    const at::Tensor key = at::empty({size()}, options.dtype(dtype()));
    const at::Tensor value = at::empty(
        {size()}, options.dtype(c10::CppTypeToScalarType<ValueType>::value));
    const auto key_data = key.data_ptr<KeyType>();
    const auto value_data = value.data_ptr<ValueType>();

    map_->retrieve_all(key_data, value_data);

    const auto perm = at::empty_like(value);
    perm.scatter_(0, value, at::arange(value.numel(), value.options()));

    return key.index_select(0, perm);
  }

  int64_t size() override { return static_cast<int64_t>(map_->size()); }

  at::ScalarType dtype() override {
    if (std::is_same<KeyType, int16_t>::value) {
      return at::kShort;
    } else if (std::is_same<KeyType, int32_t>::value) {
      return at::kInt;
    } else {
      return at::kLong;
    }
  }

  at::Device device() override { return device_; }

 private:
  std::unique_ptr<cuco::static_map<KeyType, ValueType>> map_;
  at::Device device_;
};
#endif

struct CUDAHashMap : torch::CustomClassHolder {
 public:
  CUDAHashMap(const at::Tensor& key, double load_factor = 0.5) {
#ifndef _WIN32
    at::TensorArg key_arg{key, "key", 0};
    at::CheckedFrom c{"CUDAHashMap.init"};
    at::checkDeviceType(c, key, at::DeviceType::CUDA);
    at::checkDim(c, key_arg, 1);
    at::checkContiguous(c, key_arg);

    DISPATCH_KEY(key.scalar_type(), "cuda_hash_map_init", [&] {
      map_ = std::make_unique<CUDAHashMapImpl<scalar_t>>(key, load_factor);
    });
#else
    TORCH_CHECK(false, "'CUDAHashMap' not supported on Windows");
#endif
  }

  at::Tensor get(const at::Tensor& query) {
#ifndef _WIN32
    at::TensorArg query_arg{query, "query", 0};
    at::CheckedFrom c{"CUDAHashMap.get"};
    at::checkDeviceType(c, query, at::DeviceType::CUDA);
    at::checkDim(c, query_arg, 1);
    at::checkContiguous(c, query_arg);

    return map_->get(query);
#else
    TORCH_CHECK(false, "'CUDAHashMap' not supported on Windows");
#endif
  }

  at::Tensor keys() {
#ifndef _WIN32
    return map_->keys();
#else
    TORCH_CHECK(false, "'CUDAHashMap' not supported on Windows");
#endif
  }

  int64_t size() { return map_->size(); }

  at::ScalarType dtype() { return map_->dtype(); }

  at::Device device() { return map_->device(); }

 private:
#ifndef _WIN32
  std::unique_ptr<HashMapImpl> map_;
#endif
};

}  // namespace

TORCH_LIBRARY_FRAGMENT(pyg, m) {
  m.class_<CUDAHashMap>("CUDAHashMap")
      .def(torch::init<at::Tensor&, double>())
      .def("get", &CUDAHashMap::get)
      .def("keys", &CUDAHashMap::keys)
      .def("size", &CUDAHashMap::size)
      .def("dtype", &CUDAHashMap::dtype)
      .def("device", &CUDAHashMap::device)
      .def_pickle(
          // __getstate__
          [](const c10::intrusive_ptr<CUDAHashMap>& self) -> at::Tensor {
            return self->keys();
          },
          // __setstate__
          [](const at::Tensor& state) -> c10::intrusive_ptr<CUDAHashMap> {
            return c10::make_intrusive<CUDAHashMap>(state);
          });
}

}  // namespace classes
}  // namespace pyg
